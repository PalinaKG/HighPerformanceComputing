#include "hip/hip_runtime.h"
#include "matmult_fun.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C" {
	#include <cblas.h>
	#include <omp.h>

	#define BLOCK_SIZE 11.0;
}

extern "C" {
void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
	double *d_A, *d_B, *d_C;

    /*
    printf("A\n");
    mat_print(m,k,A);
    printf("B\n");
    mat_print(k,n,B);
    */
	
	// set memory on GPU device
	hipMalloc((void **)&d_C, m * n * sizeof(double));
	hipMalloc((void **)&d_B, k * n * sizeof(double));
	hipMalloc((void **)&d_A, m * k * sizeof(double));
   
	// Copy data to device
	hipMemcpy(d_C,C, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B,B, k * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A,A, m * k * sizeof(double), hipMemcpyHostToDevice);
    
	

	// execute kernel 
	gpu1_kernel<<<1,1>>>(m,n,k,d_A,d_B,d_C);
	hipDeviceSynchronize();
	
	// transfer results from GPU device
	hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);


	// clean up data on device
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);

	}
}

__global__ void gpu1_kernel(int m,int n,int k, double *d_A, double *d_B, double *d_C){

	int i,j,l;
	double sum;

	for (l = 0; l < m; l++)
    {
        for (j = 0; j < n; j++)
        {
            sum = 0.0;
            for (i = 0; i < k; i++)
            {
                sum += d_A[l*k+i] * d_B[i*n+j]; 
            }
            d_C[l*n+j] = sum;
        }
    }
}


extern "C" {
void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
	double *d_A, *d_B, *d_C;
/*
    printf("A\n");
    mat_print(m,k,A);
    printf("B\n");
    mat_print(k,n,B);*/
	
	// set memory on GPU device
	hipMalloc((void **)&d_C, m * n * sizeof(double));
	hipMalloc((void **)&d_B, k * n * sizeof(double));
	hipMalloc((void **)&d_A, m * k * sizeof(double));
   
	// Copy data to device
	hipMemcpy(d_C,C, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B,B, k * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A,A, m * k * sizeof(double), hipMemcpyHostToDevice);
    
	

	// execute kernel 
	// <NUM_BLOCKS, THREADS PER BLOCK>
	//Number of blocks for each dimensions

	double block_size = 16.0;

	int dim_m = ceil(m/block_size);
	int dim_n = ceil(n/block_size);
	
	dim3 dimGrid(dim_m, dim_n, 1);
	dim3 dimBlock((int)block_size, (int)block_size, 1);
	gpu2_kernel<<<dimGrid, dimBlock>>>(m,n,k,d_A,d_B,d_C);
	checkCudaErrors(hipDeviceSynchronize());
	
	// transfer results from GPU device
	hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

	//mat_print(m,n,C);
	// clean up data on device
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);

	}
}

__global__ void gpu2_kernel(int m,int n,int k, double *d_A, double *d_B, double *d_C){

	int j;
	double sum=0.0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	//More threads are initialized than needed
	
	if (col < m && row < n)
	{
    		for (j = 0; j < k; j++)
    		{ 
        		sum += d_A[col*k+j] * d_B[j*n+row]; 
    		}
    		d_C[col*n+row] = sum;
	}
}


extern "C" {
void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) {
	double *d_A, *d_B, *d_C;
/*
    printf("A\n");
    mat_print(m,k,A);
    printf("B\n");
    mat_print(k,n,B);*/
	
	// set memory on GPU device
	hipMalloc((void **)&d_C, m * n * sizeof(double));
	hipMalloc((void **)&d_B, k * n * sizeof(double));
	hipMalloc((void **)&d_A, m * k * sizeof(double));
   
	// Copy data to device
	hipMemcpy(d_C,C, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B,B, k * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A,A, m * k * sizeof(double), hipMemcpyHostToDevice);
    
	

	// execute kernel 
	// <NUM_BLOCKS, THREADS PER BLOCK>
	//Number of blocks for each dimensions

	double block_size = 4.0;

	int dim_m = ceil(m/block_size);
	int dim_n = ceil(n/block_size);
	
	dim3 dimGrid(dim_m, dim_n, 1);
	dim3 dimBlock((int)block_size, (int)block_size, 1);
	gpu3_kernel<<<dimGrid, dimBlock>>>(m,n,k,d_A,d_B,d_C);
	checkCudaErrors(hipDeviceSynchronize());
	
	// transfer results from GPU device
	hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

	//mat_print(m,n,C);
	// clean up data on device
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);

	}
}

__global__ void gpu3_kernel(int m,int n,int k, double *d_A, double *d_B, double *d_C){

	int j;
	double sum1=0.0, sum2=0.0;
	int col = 2*(blockIdx.x * blockDim.x + threadIdx.x);
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	//More threads are initialized than needed
	if ((col-1) < m && row < n)
	{
    		for (j = 0; j < k; j++)
    		{ 
        		sum1 += d_A[col*k+j] * d_B[j*n+row];
			sum2 += d_A[(col+1)*k+j] * d_B[j*n+row];
    		}
    		d_C[col*n+row] = sum1;
			d_C[(col+1)*n+row] = sum2;
	
	}
	// If matrix is odd numbered, one calc remaining
	else if (col < m && row < n)
	{
    		for (j = 0; j < k; j++)
    		{ 
        		sum1 += d_A[col*k+j] * d_B[j*n+row];
    		}
    		d_C[col*n+row] = sum1;
	}

}


extern "C" {
void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C) {
	double *d_A, *d_B, *d_C;

    printf("A\n");
    mat_print(m,k,A);
    printf("B\n");
    mat_print(k,n,B);
	
	// set memory on GPU device
	hipMalloc((void **)&d_C, m * n * sizeof(double));
	hipMalloc((void **)&d_B, k * n * sizeof(double));
	hipMalloc((void **)&d_A, m * k * sizeof(double));
   
	// Copy data to device
	hipMemcpy(d_C,C, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B,B, k * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A,A, m * k * sizeof(double), hipMemcpyHostToDevice);
    
	

	// execute kernel 
	// <NUM_BLOCKS, THREADS PER BLOCK>
	//Number of blocks for each dimensions

	double block_size = 4.0;
	int nr_of_elem = 1;

	int dim_m = ceil(m/block_size);
	int dim_n = ceil(n/block_size);
	
	dim3 dimGrid(dim_m, dim_n, 1);
	dim3 dimBlock((int)block_size, (int)block_size, 1);
	gpu4_kernel<<<dimGrid, dimBlock>>>(m,n,k,d_A,d_B,d_C,nr_of_elem);
	//gpu4_kernel<<<1,1>>>(m,n,k,d_A,d_B,d_C,nr_of_elem);
	checkCudaErrors(hipDeviceSynchronize());
	
	// transfer results from GPU device
	hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

	mat_print(m,n,C);
	// clean up data on device
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);

	}
}

__global__ void gpu4_kernel(int m,int n,int k, double *d_A, double *d_B, double *d_C, int elements){
	
	int j,l;
	double sum=0.0;
	int col = elements*(blockIdx.x * blockDim.x + threadIdx.x);
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int min_el;
	if (elements < (m-col))
	{
		min_el = elements;
	}
	else {min_el = m - col;}

	//More threads are initialized than needed
	if(row<n){
	for (l = 0; l < min_el; l++)
	{
		sum = 0.0;	
    		for (j = 0; j < k; j++)
    		{ 
			sum += d_A[(col+l)*k+j] * d_B[j*n+row];
		}
		
		d_C[(col+l)*n+row] = sum;
	}
}	
}

extern "C" {
void matmult_lib(int m, int n, int k, double *A, double *B, double *C) {
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0, A, k, B, n, 0.0, C, n);
    
    //mat_print(m,n,C);
}
}


extern "C" {
void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C) {
   	double *d_A, *d_B, *d_C;
	double alpha=1.0, beta=0.0;
	// set memory on GPU device
    hipMalloc((void **)&d_C, m * n * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_A, m * k * sizeof(double));

    // Copy data to deviice
    hipMemcpy(d_C,C, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B,B, k * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_A,A, m * k * sizeof(double), hipMemcpyHostToDevice);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, k, d_B, n, &beta, d_C, n);

	hipblasDestroy(handle);
    hipMemcpy(C, d_C, m*n* sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

    //mat_print(m,n,C);
}
}


extern "C" {
void mat_print(int m, int n, double *A){

	for (int i = 0; i < m; i++)
    	{
        	for (int j=0; j < n; j++)
        	{
            	printf("%.2f     ", A[i*n+j]);
        	}
        	printf("\n");
    	}
	printf("\n\n");

}
}
